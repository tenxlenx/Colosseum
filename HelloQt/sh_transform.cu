#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "sh_transform.cuh"

#include <bitset>
#include <memory>
#include "spherical_harmonics.hpp"
#include <hipfft/hipfft.h>
#include <>
#include <filesystem>

#include <opencv2/opencv.hpp>



#ifndef HAVE_HEADER_H
#include "sh_transform_generated.cuh"
void GPU_SH::SHTransform::init_from_precalc(
                  int order = 8
                , int image_width = 128
                , int image_height = 64,
                  SphericalHarmonics* sph = nullptr)
{

    int num_coeffs = SphericalHarmonics::GetCoefficientCount(order);
    int image_size = image_height * image_width;
    int matrix_size = image_size * num_coeffs;

    const size_t rows = std::size(sh_transform_mat);
    const size_t cols = std::size(sh_transform_mat[0]);
    hipMalloc(&d_SH_Matrix_, sizeof(double) * matrix_size);
    hipMemcpy(d_SH_Matrix_, sh_transform_mat, sizeof(double) * matrix_size, hipMemcpyHostToDevice);
}
#else

void GPU_SH::SHTransform::init_from_precalc(
    int order = 8, int image_width = 128, int image_height = 64,
    SphericalHarmonics* sph = nullptr)
{
    /// Check if sh_transform_mat is defined in the header file
    // If not defined, generate it here
    if (sph == nullptr) sph = new SphericalHarmonics(image_width, image_height, order);
    sph->generate_matrix_code(order, image_width, image_height);

}

#endif



__global__ void inverse_transform_kernel(double* d_feature_matrix, int m, int n, double* d_coefficents, int n_coeffs, double* d_image)
{
    // get the thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // check if the thread index is within the bounds of the image
    if (idx < n && idy < m) {
        // initialize the sum to 0
        double sum = 0.0;

        // calculate the sum
        for (int i = 0; i < n_coeffs; i++) {
            sum += d_feature_matrix[i * n * m + idx * m + idy] * d_coefficents[i];
        }

        // store the result in the image
        d_image[idx * m + idy] = sum;
    }
}

__device__ int GetIndex(int l, int m)
{
    return l * (l + 1) + m;
}

__device__ void getLM(int index, int& L, int& M)
{
    L = (int)floor(sqrtf(index));
    M = index - (L * L + L);
}

__device__ double rotateZ(double* coefficients, int index, double rotation_degree)
{
    int L = 0;
    int M = 0;
    double coeff_n;
    double coeff_p;
    double rot_coeff = 0;
    double alpha = rotation_degree * M_PI / 180.0;
    getLM(index, L, M);

    if (M < 0) {
        coeff_n = coefficients[index];
        coeff_p = coefficients[GetIndex(L, -M)];
    }
    else {
        coeff_n = coefficients[GetIndex(L, -M)];
        coeff_p = coefficients[index];
    }
    if (M < 0) {
        rot_coeff = coeff_n * cos(abs(M) * alpha) - coeff_p * sin(abs(M) * alpha);
    }
    else {
        rot_coeff = coeff_n * sin(abs(M) * alpha) + coeff_p * cos(abs(M) * alpha);
    }
    return rot_coeff;
}

__global__ void rotateCoeffs(double* input_coeffs, double* output_coeffs, double rotation_degree, int num_lm_coeffs)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_lm_coeffs) {
        output_coeffs[index] = rotateZ(input_coeffs, index, rotation_degree);
    }
}

void GPU_SH::SHTransform::get_all_sh_rotations(double* input_coeffs, double* output_coeffs, int num_rotations)
{
    int num_lm_coeffs = sph_->GetCoefficientCount(order_);
    // Launch kernel with num_rotations threads per block, and enough blocks to cover all coefficients
    int block_size = num_rotations;
    int num_blocks = (num_lm_coeffs + block_size - 1) / block_size;
    for (int i = 0; i < num_rotations; i++) {
        double rotation_degree = 360.0 * i / num_rotations;

        rotateCoeffs<<<num_blocks, block_size>>>(input_coeffs, &output_coeffs[num_lm_coeffs * i], rotation_degree, num_lm_coeffs);
        hipDeviceSynchronize();
        // process the rotated coefficients here, e.g. save them to a file
    }
}

template <typename T>
std::vector<T> flatten(const std::vector<std::vector<T>>& orig)
{
    std::vector<T> ret;
    for (const auto& v : orig)
        ret.insert(ret.end(), v.begin(), v.end());
    return ret;
}

std::vector<double> GPU_SH::SHTransform::transform(cv::Mat& img)
{
    //calculates coeffs
    tf(img);
    int numElements = sh_count();
    double* tempHostArray = new double[numElements];
    hipMemcpy(tempHostArray, d_coeffs, numElements * sizeof(double), hipMemcpyDeviceToHost);
    std::vector<double> hostVector(numElements);
    hostVector.assign(tempHostArray, tempHostArray + numElements);
    delete[] tempHostArray;
    return hostVector;
}

void GPU_SH::SHTransform::tf(cv::Mat& img)
{
    int m = sph_->GetCoefficientCount(order_);
    int n = img.rows * img.cols;

    transfromSH(m, n, d_SH_Matrix_, img, d_coeffs);
}

cv::Mat GPU_SH::SHTransform::inverse_transform(std::vector<double> coefficients)
{
    int num_coeffs = sph_->GetCoefficientCount(order_);
    int m = image_height_;
    int n = image_width_;

    hipMemcpy(d_coeffs, coefficients.data(), num_coeffs * sizeof(double), hipMemcpyHostToDevice);

    double* d_image;
    hipMalloc(&d_image, n * m * sizeof(double));

    inverse_transform(d_SH_Matrix_, m, n, d_coeffs, num_coeffs, d_image);

    cv::Mat result_image(m, n, CV_64F);
    hipMemcpy(result_image.ptr<double>(), d_image, n * m * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_image);

    return result_image;
}

GPU_SH::SHTransform::~SHTransform()
{
    hipFree(d_coeffs);
    hipFree(d_SH_Matrix_);
    hipFree(d_current_image_);
    hipblasDestroy(handle_);
}

int GPU_SH::SHTransform::sh_count()
{
    return sph_->GetCoefficientCount(order_);
}

void GPU_SH::SHTransform::transfromSH(int m, int n, double* d_A, cv::Mat& current_image, double* d_Y)
{
    // If input image is not CV_64F, convert to CV_64F
    if (current_image.type() != CV_64F) {
        current_image.convertTo(current_image, CV_64F);
    }

    // Multiply the feature matrix (d_A) with the image (1d flattened) to get the coefficients
    hipMemcpy(d_current_image_, current_image.ptr<double>(), current_image.rows * current_image.cols * sizeof(double), hipMemcpyHostToDevice);

    double alpha = 1.0;
    double beta = 0.0;

    hipblasDgemv(handle_, HIPBLAS_OP_T, n, m, &alpha, d_A, n, d_current_image_, 1, &beta, d_Y, 1);

    hipDeviceSynchronize();
}

int GPU_SH::SHTransform::get_index(int L, int m)
{
    return sph_->GetIndex(L, m);
}

int GPU_SH::SHTransform::get_order()
{
    return order_;
}

double GPU_SH::SHTransform::factorial(int num)
{
    return sph_->factorial(num);
}




GPU_SH::SHTransform::SHTransform(int image_width, int image_height, int order, hipblasHandle_t handle = nullptr)
    : d_SH_Matrix_(nullptr)
    , image_height_(image_height)
    , image_width_(image_width)
    , order_(order)
{
    hipMalloc((void**)&d_current_image_, image_width_ * image_height_ * sizeof(double));
    sph_ = std::make_unique<SphericalHarmonics>(image_width_, image_height_, order_);
    auto trans_mat = sph_->transform_matrix();

    // if generated header exists, we use the generated file otherwise, we generate one
    init_from_precalc(order, image_width, image_height, sph_.get());
    

    if (handle == nullptr) {
        hipblasCreate(&handle_);
    }
    else {
        handle_ = handle;
    }

    // uploading matrix to gpu
    int num_coeffs = sph_->GetCoefficientCount(order_);
    auto mat_size = image_height_ * image_width_ * num_coeffs;
    hipMalloc(&d_coeffs, num_coeffs * sizeof(double));
    hipMalloc(&d_SH_Matrix_, mat_size * sizeof(double));
    hipMemcpy(d_SH_Matrix_, flatten(trans_mat).data(), mat_size * sizeof(double), hipMemcpyHostToDevice);
}

// function call
void GPU_SH::SHTransform::inverse_transform(double* d_feature_matrix, int m, int n, double* d_coefficents, int n_coeffs, double* d_image)
{
    // calculate the number of blocks and threads
    int block_size = 32;
    int num_blocks_x = (n + block_size - 1) / block_size;
    int num_blocks_y = (m + block_size - 1) / block_size;

    // launch the kernel
    inverse_transform_kernel<<<num_blocks_x, num_blocks_y>>>(d_feature_matrix, m, n, d_coefficents, n_coeffs, d_image);
}