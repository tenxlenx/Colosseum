#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "common/common_utils/StrictMode.hpp"
STRICT_MODE_OFF
#ifndef RPCLIB_MSGPACK
#define RPCLIB_MSGPACK clmdep_msgpack
#endif // !RPCLIB_MSGPACK
#include "rpc/rpc_error.h"
STRICT_MODE_ON

#include "vehicles/car/api/CarRpcLibClient.hpp"
#include "common/common_utils/FileSystem.hpp"
#include <iostream>
#include <chrono>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/opencv.hpp>
#include <cmath>
#include <chrono>
#include <iomanip>
#include <sstream>
#include <filesystem>
#include <thread>
#include <mutex>
#include <queue>
#include <condition_variable>
#include "gpu_dct.h"


constexpr double kSemiMajorAxis = 6378137.0;
constexpr double kSemiMinorAxis = 6356752.3142;
constexpr double kEccentricity = 8.1819190842622e-2;

struct ImageWriteTask
{
    std::string file_path;
    cv::Mat image;
};




struct GpsCoordinate
{
    double latitude;
    double longitude;
    double altitude;
};

struct EcefCoordinate
{
    double x;
    double y;
    double z;
};

struct CartesianCoordinate
{
    double x;
    double y;
    double z;
};

EcefCoordinate gpsToEcef(const GpsCoordinate& gps)
{
    double lat_rad = gps.latitude * M_PI / 180.0;
    double lon_rad = gps.longitude * M_PI / 180.0;
    double N = kSemiMajorAxis / std::sqrt(1 - kEccentricity * kEccentricity * std::sin(lat_rad) * std::sin(lat_rad));
    double x = (N + gps.altitude) * std::cos(lat_rad) * std::cos(lon_rad);
    double y = (N + gps.altitude) * std::cos(lat_rad) * std::sin(lon_rad);
    double z = ((1 - kEccentricity * kEccentricity) * N + gps.altitude) * std::sin(lat_rad);
    return { x, y, z };
}

CartesianCoordinate ecefToEnu(const EcefCoordinate& ecef, const EcefCoordinate& ref_ecef, const GpsCoordinate& ref_gps)
{
    double lat_rad = ref_gps.latitude * M_PI / 180.0;
    double lon_rad = ref_gps.longitude * M_PI / 180.0;

    double dx = ecef.x - ref_ecef.x;
    double dy = ecef.y - ref_ecef.y;
    double dz = ecef.z - ref_ecef.z;

    double x = -std::sin(lon_rad) * dx + std::cos(lon_rad) * dy;
    double y = -std::sin(lat_rad) * std::cos(lon_rad) * dx - std::sin(lat_rad) * std::sin(lon_rad) * dy + std::cos(lat_rad) * dz;
    double z = std::cos(lat_rad) * std::cos(lon_rad) * dx + std::cos(lat_rad) * std::sin(lon_rad) * dy + std::sin(lat_rad) * dz;

    return { x, y, z };
}

CartesianCoordinate gpsToEnu(const GpsCoordinate& gps, const GpsCoordinate& ref_gps)
{
    EcefCoordinate ecef = gpsToEcef(gps);
    EcefCoordinate ref_ecef = gpsToEcef(ref_gps);
    return ecefToEnu(ecef, ref_ecef, ref_gps);
}

struct EulerAngle
{
    double roll;
    double pitch;
    double yaw;
};

EulerAngle quaternionToEuler(const msr::airlib::Quaternionr& q)
{
    double roll = std::atan2(2.0 * (q.w() * q.x() + q.y() * q.z()), 1.0 - 2.0 * (q.x() * q.x() + q.y() * q.y()));
    double pitch = std::asin(2.0 * (q.w() * q.y() - q.z() * q.x()));
    double yaw = std::atan2(2.0 * (q.w() * q.z() + q.x() * q.y()), 1.0 - 2.0 * (q.y() * q.y() + q.z() * q.z()));

    return { roll, pitch, yaw };
}

std::string createUniqueFolder()
{
    auto now = std::chrono::system_clock::now();
    auto seconds_since_epoch = std::chrono::duration_cast<std::chrono::seconds>(now.time_since_epoch()).count();

    std::stringstream folder_name_ss;
    folder_name_ss << "data_" << seconds_since_epoch;

    std::string folder_name = folder_name_ss.str();
    std::filesystem::create_directories(folder_name);

    return folder_name;
}

void image_writer_thread_func(
    std::queue<ImageWriteTask>& task_queue,
    std::mutex& task_queue_mutex,
    std::condition_variable& task_queue_cv,
    bool& stop_thread)
{
    while (true) {
        std::unique_lock<std::mutex> lock(task_queue_mutex);
        task_queue_cv.wait(lock, [&] { return !task_queue.empty() || stop_thread; });

        if (stop_thread && task_queue.empty()) {
            break;
        }

        ImageWriteTask task = task_queue.front();
        task_queue.pop();
        lock.unlock();

        cv::imwrite(task.file_path, task.image);
    }
}


int main()
{
    using namespace msr::airlib;
    bool isCarMode = false;

    std::cout << "Make sure settings.json has \"SimMode\"=\"Car\" at root. Press Enter to continue." << std::endl;
    std::cin.get();

    msr::airlib::CarRpcLibClient client("127.0.0.1");

    typedef ImageCaptureBase::ImageRequest ImageRequest;
    typedef ImageCaptureBase::ImageResponse ImageResponse;
    typedef ImageCaptureBase::ImageType ImageType;
    typedef common_utils::FileSystem FileSystem;

    auto vehicles = client.listVehicles();
    for (auto v : vehicles) {
        std::cout << "VEHICLE: " << v << std::endl;
    }

    if (vehicles[0] == "drone_flight") {
        isCarMode = false;
    }
    else {
        isCarMode = true;
    }

    std::string data_folder = createUniqueFolder();
    std::ofstream data_file;
    data_file.open(data_folder + "/sensor_data.csv");
    if (isCarMode) {
        data_file << "timestamp,vehicle_type,x,y,z,yaw,pitch,roll,speed,steering,image_name" << std::endl;
    }
    else {
        data_file << "timestamp,vehicle_type,x,y,z,yaw,pitch,roll,speed,steering,image_name" << std::endl;
    }

    GpsCoordinate ref_gps;
    bool ref_gps_initialized = false;

     // Create the image writer thread
    std::queue<ImageWriteTask>image_write_task_queue;
    std::mutex image_write_task_queue_mutex;
    std::condition_variable image_write_task_queue_cv;
    bool stop_image_writer_thread = false;

    std::thread image_writer_thread(
        image_writer_thread_func,
        std::ref(image_write_task_queue),
        std::ref(image_write_task_queue_mutex),
        std::ref(image_write_task_queue_cv),
        std::ref(stop_image_writer_thread));

    try {
        client.confirmConnection();

        while (1) {
            vector<ImageRequest> request = { ImageRequest("panorama", ImageType::CubeScene, false, true) };
            const vector<ImageResponse>& response = client.simGetImages(request);
            cv::Mat h_result = cv::imdecode(response[0].image_data_uint8, 1);

            auto gps_data = client.getGpsData();
            auto imu_data = client.getImuData();
            EulerAngle euler_angles = quaternionToEuler(imu_data.orientation);
            double heading = euler_angles.yaw * 180.0 / M_PI; // Convert radians to degrees
            double pitch = euler_angles.pitch * 180.0 / M_PI;
            double roll = euler_angles.pitch * 180 / M_PI;

            if (!ref_gps_initialized) {
                ref_gps = GpsCoordinate{ gps_data.gnss.geo_point.latitude, gps_data.gnss.geo_point.longitude, gps_data.gnss.geo_point.altitude };
                ref_gps_initialized = true;
            }

            GpsCoordinate current_gps{ gps_data.gnss.geo_point.latitude, gps_data.gnss.geo_point.longitude, gps_data.gnss.geo_point.altitude };
            CartesianCoordinate local_xyz = gpsToEnu(current_gps, ref_gps);

            std::string image_name = data_folder + "/panorama_" + std::to_string(response[0].time_stamp) + ".jpg";
            // Add the image write task to the queue
            {
                std::lock_guard<std::mutex> lock(image_write_task_queue_mutex);
                image_write_task_queue.push(ImageWriteTask{ image_name, h_result });
            }
            image_write_task_queue_cv.notify_one();

            if (isCarMode) {
                auto car_controls = client.getCarControls();
                auto car_state = client.getCarState();
                data_file << response[0].time_stamp << ","
                          << "CAR" << ","                                            
                          << local_xyz.x << ","
                          << local_xyz.y << ","
                          << local_xyz.z << ","
                          << heading << ","
                          << pitch << ","
                          << roll << ","
                          << car_state.speed << ","
                          << car_controls.steering << ","
                          << image_name << std::endl;
            }
            else {
                data_file << response[0].time_stamp << ","
                          << local_xyz.x << ","
                          << local_xyz.y << ","
                          << local_xyz.z << ","
                          << heading << ","
                          << pitch << ","
                          << roll << ","
                          << 0 << ","
                          << 0 << ","
                          << image_name << std::endl;
            }
            cv::imshow("pano", h_result);
            cv::waitKey(1);

            
        }
    }
    catch (rpc::rpc_error& e) {
        std::string msg = e.get_error().as<std::string>();
        std::cout << "Exception raised by the API, something went wrong." << std::endl
                  << msg << std::endl;
        std::cin.get();
    }

    data_file.close();
    return 0;
}
